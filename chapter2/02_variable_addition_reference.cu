//
// Created by Administrator on 2021/12/28.
//
#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdio>

// kernel function to add two variables, parameter are passed by reference
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c){
    *d_c = *d_a + *d_b;
}

int main(void){
    // define host variables
    int h_a, h_b, h_c;
    // define device pointers
    int *d_a, *d_b, *d_c;
    // initializing host varibles
    h_a = 1;
    h_b = 4;

    // allocating memory for devices pointers
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // copy value of host variable in device memory
    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
    // call kernel with one thread and one block with parameter passed by reference
    gpuAdd<<<1,1>>>(d_a, d_b, d_c);

    //copying result from device to host
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("passing parameter by reference output: %d + %d = %d\n", h_a, h_b, h_c);

    // free up memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}














