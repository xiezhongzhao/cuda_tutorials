#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 5

__global__ void gpuSquare(float *d_in, float *d_out){
    int tid = threadIdx.x;
    float temp = d_in[tid];
    d_out[tid] = temp * temp;
}

int main(void){
    float h_in[N], h_out[N];
    float *d_in, *d_out;

    // allocate the memory on the cpu
    hipMalloc((void**)&d_in, N*sizeof(float));
    hipMalloc((void**)&d_out, N*sizeof(float));
    for(int i=0; i<N; i++){
        h_in[i] = i;
    }

    // copy array from host to device
    hipMemcpy(d_in, h_in, N*sizeof(float), hipMemcpyHostToDevice);
    gpuSquare<<<1, N>>>(d_in, d_out);
    hipMemcpy(h_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        printf("the square of %f is %f\n", h_in[i], h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}

