#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 50

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c){
    int tid = blockIdx.x; // handle the data at the index
    if(tid < N)
        d_c[tid] = d_a[tid] + d_b[tid];
}

int main(void){

    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    // allocate the memory
    hipMalloc((void**)&d_a, N*sizeof(int));
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&d_c, N*sizeof(int));
    for(int i=0; i<N; i++){
        h_a[i] = 2 * i*i;
        h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);
    gpuAdd<<<N, 1>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        printf("The sum of %d element is %d + %d = %d\n", i, h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "hello world " << std::endl;
    return 0;
}




























