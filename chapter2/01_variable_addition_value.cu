#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>


// define of kernel function to add two variables
__global__ void gpuAdd(int d_a, int d_b, int *d_c){
    *d_c = d_a + d_b;
}

int main() {
    //define host variable to store answer
    int h_c;
    //define device pointer
    int *d_c;

    //allocating memory for device pointer
    hipMalloc((void**)&d_c, sizeof(int));
    //kernel call by passing 1 and 4 as inputs and storing answer in d_c
    //<<<1,1>>> means 1 block is executed with 1 thread per block
    gpuAdd<<<1,1>>>(1, 4, d_c);
    //copy result from device memory to host memory
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("1+4 = %d\n", h_c);
    hipFree(d_c);

    return 0;
}
























