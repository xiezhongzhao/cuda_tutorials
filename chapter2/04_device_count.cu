//
// Created by Administrator on 2021/12/28.
//
#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void){

    int device_out = 0;
    hipGetDeviceCount(&device_out);
    // this function returns count of number of cuda enable devices and
    // 0 if there are no cuda capable devices
    if(device_out == 0){
        printf("There are no available devices that support cuda \n");
    }else{
        printf("detected %d cuda capable devices\n", device_out);
    }
    return 0;
}





