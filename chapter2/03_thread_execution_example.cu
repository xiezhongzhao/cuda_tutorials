//
// Created by Administrator on 2021/12/28.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__ void myfirstkernel(void){
    //blockIdx.x gives the block number of current kernel
    printf("hello!!! I am thread in block: %d\n", blockIdx.x);
}

int main(void){
    // a kernel call with 16 blocks and 1 thread per block
    myfirstkernel<<<16,1>>>();
    // function used for waiting for all kernels to finish
    hipDeviceSynchronize();
    printf("All threads are finished!\n");
    return 0;
}



