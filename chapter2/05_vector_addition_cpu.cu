//
// Created by Administrator on 2021/12/28.
//


#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
// define number of elements in array
#define N 5

//defining vector addition function for CPU
void cpuAdd(int *h_a, int *h_b, int *h_c){
    int tid = 0;
    while(tid < N){
        h_c[tid] = h_a[tid] + h_b[tid];
        tid += 1;
    }
}

int main(void){
    int h_a[N]={0}, h_b[N]={0}, h_c[N]={0};

    // initializing two arrays for addition
    for(int i=0; i<N; i++){
        h_a[i] = 2 * i*i;
        h_b[i] = i;
    }
    // call CPU function for vector addition
    cpuAdd(h_a, h_b, h_c);
    // printing answer
    printf("vector addition on CPU\n");
    for(int i=0; i<N; i++){
        printf("the sum of %d element is %d + %d = %d\n", i, h_a, h_b, h_c);
    }
    return 0;
}














