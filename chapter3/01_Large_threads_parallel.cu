#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 50000

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    while(tid < N){
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void){
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, N*sizeof(int));
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&d_c, N*sizeof(int));
    for(int i=0; i<N; i++){
        h_a[i] = 2 * i*i;
        h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);
    gpuAdd<<<512,512>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    int Correct = 1;
    printf("Vector addition on GPU \n");
    //Printing result on console
    for (int i = 0; i < N; i++) {
        if ((h_a[i] + h_b[i] != h_c[i])){
            Correct = 0;
        }

    }
    if (Correct == 1){
        printf("GPU has computed Sum Correctly\n");
    }
    else{
        printf("There is an Error in GPU Computation\n");
    }
    //Free up memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}




















