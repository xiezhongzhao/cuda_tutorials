
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpu_shared_memory(float *d_a)
{
    // Defining local variables which are private to each thread
    int i, index = threadIdx.x;
    float average, sum = 0.0f;
    //Define shared memory
    __shared__ float sh_arr[10];

    sh_arr[index] = d_a[index];
    __syncthreads();    // This ensures all the writes to shared memory have completed
    for (i = 0; i<= index; i++){
        sum += sh_arr[i];
    }
    average = sum / (index + 1.0f);
    d_a[index] = average;
    sh_arr[index] = average;
}

int main(int argc, char **argv)
{
    //Define Host Array
    float h_a[10];
    //Define Device Pointer
    float *d_a;

    for (int i = 0; i < 10; i++) {
        h_a[i] = i;
    }
    // allocate global memory on the device
    hipMalloc((void **)&d_a, sizeof(float) * 10);
    // now copy data from host memory  to device memory
    hipMemcpy((void *)d_a, (void *)h_a, sizeof(float) * 10, hipMemcpyHostToDevice);

    gpu_shared_memory <<<1, 10 >>>(d_a);
    // copy the modified array back to the host memory
    hipMemcpy((void *)h_a, (void *)d_a, sizeof(float) * 10, hipMemcpyDeviceToHost);
    printf("Use of Shared Memory on GPU:  \n");
    //Printing result on console
    for (int i = 0; i < 10; i++) {
        printf("The running average after %d element is %f \n", i, h_a[i]);
    }
    return 0;
}

