
#include <hip/hip_runtime.h>
#include <cstdio>
#define N 5

__global__ void gpu_local_memory(int d_in){
    int t_local;
    t_local = d_in * threadIdx.x;
    printf("value of local variable in current thread is %d \n", t_local);
}

int main(int argc, char **argv){
    printf("use of local memory on GPU \n");
    gpu_local_memory<<<1,N>>>(5);
    hipDeviceSynchronize();
    return 0;
}


